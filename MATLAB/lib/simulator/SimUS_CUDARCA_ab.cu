
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <hip/hip_complex.h>

using namespace std;

__constant__ float  PI = (float)(3.14159265359);

__constant__ float  Ar[4] = {0.187, 0.288, 0.187, 0.288};
__constant__ float  Ai[4] = {0.275 , -1.954, -0.275, 1.954};
__constant__ float  Br[4] = {4.558, 8.598,  4.558, 8.598};
__constant__ float  Bi[4] = {-25.59, -7.924, 25.59, 7.924};


// Device Functions

// sinc fonction
__device__ float Sinc(float x)
{
    if(fabsf(x)>0)
    {
        if(fabsf(x)<(PI))
        {
            return sinf(x)/(x);
        }else{
            return 0;
        }
    }else{
        return 1;
    }
}

// Complex exponential
__device__ hipFloatComplex cuCexpf(hipFloatComplex x)
{
    float realx = hipCrealf(x);
    float imagx = hipCimagf(x);
    hipFloatComplex res = make_hipFloatComplex( expf(realx)*cosf(imagx), expf(realx)*sinf(imagx) );
    return res ;
}

// cuCsinc fonction
__device__ hipFloatComplex cuCSinc(hipFloatComplex x)
{
    if(hipCabsf(x)>0)
    {
    //    return sinf(x)/(x);
    //    return make_cuFloatComplex(sinf(PI*cuCabsf(x))/(PI*cuCabsf(x)), 0); // Warning
        if(hipCabsf(x)<(PI))
        {
        hipFloatComplex Den      = make_hipFloatComplex( 2*hipCimagf(x), -2*hipCrealf(x));
        hipFloatComplex Arg_p    = make_hipFloatComplex( hipCimagf(x), -hipCrealf(x));
        hipFloatComplex Arg_m    = make_hipFloatComplex( -hipCimagf(x),hipCrealf(x));

        hipFloatComplex cuSincf  = hipCdivf( hipCsubf( cuCexpf(Arg_p), cuCexpf(Arg_m)), Den);

        return cuSincf; //

        }else{
        return make_hipFloatComplex( 0, 0);
        }

    }else{
        return make_hipFloatComplex( 1, 0);
    }
}

// cuCSqrt
__device__ hipFloatComplex cuCsqrtf(hipFloatComplex x)
{
    float realx = hipCrealf(x);
    float imagx = hipCimagf(x);
    float ampx  = sqrtf(realx*realx + imagx*imagx);
    float aglx  = atan2f(imagx, realx);
    hipFloatComplex amp = make_hipFloatComplex( sqrtf(ampx), 0);
    hipFloatComplex agl = make_hipFloatComplex( 0, fdividef(aglx, 2.0f));
    hipFloatComplex res = hipCmulf( amp, cuCexpf( agl ) );
    return res;
}

// Green fonction
__device__ hipFloatComplex Green(float r, float kWaveNumber)
{
    float reGreen = fdividef( cosf(kWaveNumber*r), r);
    float imGreen = fdividef( sinf(kWaveNumber*r), r);
    return make_hipFloatComplex( reGreen, imGreen );
}

// cuCGreen fonction
__device__ hipFloatComplex cuCGreen(float r, hipFloatComplex kWaveNumber)
{
    hipFloatComplex rC = make_hipFloatComplex( r, 0 );
    hipFloatComplex I  = make_hipFloatComplex(0, 1.0f);
    hipFloatComplex Im = make_hipFloatComplex(0, -1.0f);

    return hipCdivf( cuCexpf( hipCmulf( hipCmulf(I,kWaveNumber), rC) ), rC); // exp(-kr)/r

}

// Directivity flat
__device__ hipFloatComplex DirFlat(float theta, float kWaveNumber, float width, float baffle)
{
    float cosTheta = cosf( theta );
    float sinTheta = sinf( theta );

    float DirFlat  = Sinc( fdividef( sinTheta, 2) * kWaveNumber * width);
    if(baffle==1)
    {
       DirFlat = DirFlat * cosTheta;
    }
    return make_hipFloatComplex( DirFlat, 0 );
}

// Directivity flat
__device__ hipFloatComplex cuCDirFlat(float theta, hipFloatComplex kWaveNumber, float width, float baffle)
{
//    float cosTheta = cosf( theta );
    float sinTheta = sinf( theta );

//     float DirFlat  = Sinc( fdividef( sinTheta, 2) * kWaveNumber * width);
    hipFloatComplex arg = make_hipFloatComplex( fdividef( sinTheta, 2) * width, 0);
    hipFloatComplex DirFlat  = cuCSinc( hipCmulf(arg, kWaveNumber) );

    if(baffle==1)
    {
       hipFloatComplex cosTheta = make_hipFloatComplex(cosf( theta ), 0);
       DirFlat = hipCmulf( DirFlat , cosTheta);
    }
    return DirFlat;
}

// Directivity Curved
__device__ hipFloatComplex DirCurv(float r, float y, float kWaveNumber, float height, float rfocus)
{

    float r_diff_norm = fdividef(r - rfocus, r * rfocus);
    hipFloatComplex height2 = make_hipFloatComplex(height*height,0);

    hipFloatComplex beta = make_hipFloatComplex(0, -kWaveNumber * fdividef(y, r) );
    hipFloatComplex gama = make_hipFloatComplex(0, kWaveNumber * fdividef(y*y, 2*r) );

    hipFloatComplex alpha = make_hipFloatComplex(0, .5*kWaveNumber*r_diff_norm);

    hipFloatComplex alpha_g;
    hipFloatComplex exp_g;
    hipFloatComplex sqrt_g;

    hipFloatComplex DirCurv = make_hipFloatComplex(0, 0);

    for(int ig = 0; ig<4; ig++)
    {
        hipFloatComplex Ag = make_hipFloatComplex(Ar[ig], Ai[ig]);
        hipFloatComplex Bg = make_hipFloatComplex(Br[ig], Bi[ig]);

        alpha_g = hipCaddf( hipCdivf(Bg, height2), alpha);

        exp_g = cuCexpf( hipCaddf( hipCdivf( hipCmulf(beta, beta),
                                           hipCmulf(alpha_g, make_hipFloatComplex(4.0f, 0) ) ), gama) );
        sqrt_g = cuCsqrtf( hipCdivf( make_hipFloatComplex(PI, 0), alpha_g) );

        DirCurv = hipCaddf( DirCurv, hipCmulf(Ag, hipCmulf(sqrt_g, exp_g) ) );
    }

    return DirCurv;
}

// Directivity Curved
__device__ hipFloatComplex cuCDirCurv(float r, float y, hipFloatComplex kWaveNumber, float height, float rfocus)
{

    hipFloatComplex r_diff_norm_2 = make_hipFloatComplex( fdividef(r - rfocus, r * rfocus) * 0.5f, 0);
    hipFloatComplex height2 = make_hipFloatComplex(height*height,0);

    hipFloatComplex y_r  = make_hipFloatComplex(fdividef(y, r), 0);
    hipFloatComplex y2_2r  = make_hipFloatComplex(fdividef(y*y, 2*r), 0);
    hipFloatComplex I    = make_hipFloatComplex(0, 1.0f);
    hipFloatComplex Im   = make_hipFloatComplex(0, -1.0f);

    hipFloatComplex beta = hipCmulf( hipCmulf(Im, kWaveNumber), y_r);
    hipFloatComplex gama = hipCmulf( hipCmulf(I, kWaveNumber), y2_2r);

    hipFloatComplex alpha = hipCmulf( hipCmulf(I, kWaveNumber), r_diff_norm_2);

    hipFloatComplex alpha_g;
    hipFloatComplex exp_g;
    hipFloatComplex sqrt_g;

    hipFloatComplex DirCurv = make_hipFloatComplex(0, 0);

    for(int ig = 0; ig<4; ig++)
    {
        hipFloatComplex Ag = make_hipFloatComplex(Ar[ig], Ai[ig]);
        hipFloatComplex Bg = make_hipFloatComplex(Br[ig], Bi[ig]);

        alpha_g = hipCaddf( hipCdivf(Bg, height2), alpha);

        exp_g = cuCexpf( hipCaddf( hipCdivf( hipCmulf(beta, beta),
                                           hipCmulf(alpha_g, make_hipFloatComplex(4.0f, 0) ) ), gama) );
        sqrt_g = cuCsqrtf( hipCdivf( make_hipFloatComplex(PI, 0), alpha_g) );

        DirCurv = hipCaddf( DirCurv, hipCmulf(Ag, hipCmulf(sqrt_g, exp_g) ) );
    }

    return DirCurv;
}

// TXfield (kWaveNumber, r_scat, event)
__global__ void TXfield(hipFloatComplex *TX,
        const float *r_scat,
        const float *r_ele,
        const hipFloatComplex *kWaveNumber,
        const hipFloatComplex *PULSE,
        const hipFloatComplex *IR,
        const float *Delay,
        const float *Apod,
        const float width,
        const float height,
        const float rfocus,
        const float baffle,
        const float lensCorrection,
        const float SpeedOfSound,
        const int NscatPerEvent,
        const int Nele,
        const int Nf,
        const int Ntx,
        const int Nsubele)
{
    int ik      = blockIdx.x * blockDim.x + threadIdx.x;   //kwaveIndex index
    int iscat	= blockIdx.y * blockDim.y + threadIdx.y;   //scatter index
    int itx     = blockIdx.z * blockDim.z + threadIdx.z;   //transmit index

    if(ik<Nf)
    {
    if(iscat<NscatPerEvent)
    {
    if(itx<Ntx)
    {

        hipFloatComplex TXtmp = make_hipFloatComplex(0, 0);

        // Output index
        int idv         = (ik + iscat*Nf + itx*Nf*NscatPerEvent);  //

        // Scatterer Position
        float Xscat     = r_scat[iscat + 3*NscatPerEvent*itx];
        float Yscat     = r_scat[iscat + NscatPerEvent + 3*NscatPerEvent*itx];
        float Zscat     = r_scat[iscat + 2*NscatPerEvent + 3*NscatPerEvent*itx];

        // Wavenumber
        hipFloatComplex k = kWaveNumber[ik];
        float w = hipCrealf(kWaveNumber[ik]) * SpeedOfSound;

        hipFloatComplex TXele, s_tx;
        float Xele, Yele, Zele, Azele, Elele, r, az, el;

        for(int iele = 0; iele < Nele; iele++)
        {
            for(int isub = 0; isub < Nsubele; isub++)
            {
                // Element Position
                Xele      = r_ele[iele + isub*5*Nele];
                Yele      = r_ele[iele + Nele + isub*5*Nele];
                Zele      = r_ele[iele + 2*Nele + isub*5*Nele];
                Azele     = r_ele[iele + 3*Nele + isub*5*Nele];
                Elele     = r_ele[iele + 4*Nele + isub*5*Nele];

                // Distance To Element
                r         = norm3df(Xscat - Xele, Yscat - Yele, Zscat - Zele);
                az        = asinf( fdividef(Xscat - Xele, r)) - Azele;
                el        = asinf( fdividef(Yscat - Yele, r)) - Elele;

                // Transmit Wavefield
                TXele     = cuCGreen(r, k);
                TXele     = hipCmulf(TXele, cuCDirFlat(az, k, width, baffle) );
                if(rfocus>10.0f)
                {
                TXele	= hipCmulf(TXele, cuCDirFlat(el, k, width, baffle) );
                }else{
                TXele	= hipCmulf(TXele, cuCDirCurv(r, r*sinf(el), k, height, rfocus) );
                }
                TXele     = hipCmulf(TXele, cuCexpf( make_hipFloatComplex(0, w*lensCorrection) ) );
                TXele     = hipCmulf(TXele, hipCmulf( hipConjf( PULSE[ik] ), hipConjf( IR[ik] ))  );

                // Transmit Wave Front
                s_tx      = hipCmulf( make_hipFloatComplex( Apod[iele + itx*Nele], 0),
                                     cuCexpf( make_hipFloatComplex(0, w*Delay[iele + itx*Nele]) ) );
                TXele     = hipCmulf( TXele, s_tx);

                // Sum the contribution of every element
                TXtmp     = hipCaddf( TXtmp, TXele);
            }
        }
        TX[idv] = TXtmp;
    }
    }
    }
}

// RXfield (kWaveNumber, r_ele, iscat)
__global__ void RXfield(hipFloatComplex *RX,
        const float *r_scat,
        const float *r_ele,
        const hipFloatComplex *kWaveNumber,
        const hipFloatComplex *IR,
        const float width,
        const float height,
        const float rfocus,
        const float baffle,
        const float lensCorrection,
        const float SpeedOfSound,
        const int Nscat,
        const int Nele,
        const int Nf,
        const int Nsubele)
{
    int ik      = blockIdx.x * blockDim.x + threadIdx.x;   //frequency index
    int iele    = blockIdx.y * blockDim.y + threadIdx.y;   //element index
    int iscat   = blockIdx.z * blockDim.z + threadIdx.z;   //scatter index

    if(iscat<Nscat)
    {
    if(iele<Nele)
    {
    if(ik<Nf)
    {
        // Allocate output
        hipFloatComplex RXout = make_hipFloatComplex(0, 0);;
        // Output index
        int idv         = (ik + iele*Nf + iscat*Nf*Nele);  //

        // Scatterer Position
        float Xscat     = r_scat[iscat];
        float Yscat     = r_scat[iscat + Nscat];
        float Zscat     = r_scat[iscat + 2*Nscat];

        // Wavenumber
        hipFloatComplex k = kWaveNumber[ik];
        float w = hipCrealf(kWaveNumber[ik]) * SpeedOfSound;

        float Xele, Yele, Zele, Azele, Elele, r, az, el;

        hipFloatComplex RXtmp;
        for(int isub = 0; isub < Nsubele; isub++)
        {
            // Element Position
            Xele      = r_ele[iele + isub*5*Nele];
            Yele      = r_ele[iele + Nele + isub*5*Nele];
            Zele      = r_ele[iele + 2*Nele + isub*5*Nele];
            Azele     = r_ele[iele + 3*Nele + isub*5*Nele];
            Elele     = r_ele[iele + 4*Nele + isub*5*Nele];

            // Distance To Element
            r         = norm3df(Xscat - Xele, Yscat - Yele, Zscat - Zele);
            az        = asinf( fdividef(Xscat - Xele, r)) - Azele;
            el        = asinf( fdividef(Yscat - Yele, r)) - Elele;

            // Transmit Wavefield
            RXtmp     = cuCGreen(r, k);
            RXtmp     = hipCmulf(RXtmp, cuCDirFlat(az, k, width, baffle) );
            if(rfocus>10.0f)
            {
            RXtmp	= hipCmulf(RXtmp, cuCDirFlat(el, k, width, baffle) );
            }else{
            RXtmp	= hipCmulf(RXtmp, cuCDirCurv(r, r*sinf(el), k, height, rfocus) );
            }
            RXtmp     = hipCmulf(RXtmp, cuCexpf( make_hipFloatComplex(0, w*lensCorrection) ) );
            RXtmp     = hipCmulf(RXtmp, IR[ik] );
            
            // Sum the contribution of every element
            RXout     = hipCaddf( RXout, RXtmp);

        }
        // Return output
        RX[idv]	  = RXout;
    }
    }
    }
}